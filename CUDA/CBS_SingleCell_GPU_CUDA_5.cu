#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <chrono>
#include "hip/hip_runtime_api.h"

using namespace std;

double errorcalcualtion(hipDoubleComplex* M, hipDoubleComplex* Q, const int Ncn1, const int Kcn1);
__global__ void fftshift_g(hipDoubleComplex* M, const int N1, const int N2, const int Kcn);
__global__ void absorbinglayer_g(double* mask11, const int Ncn1, const int Kcn1, const int PML1, double epsilon1, double dx1);
__global__ void applyboundarycondition_g(hipDoubleComplex* Mfn, double* ABL1, const int Ncn1);
__global__ void multiply_g(hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* C, const int Ncn1);
__global__ void s_multiply_g(hipDoubleComplex* A, hipDoubleComplex B, hipDoubleComplex* C, const int Ncn1);
__global__ void sum_g(hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* C, const int Ncn1);
__global__ void sub_g(hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* C, const int Ncn1);
__global__ void s_divide_g(hipDoubleComplex* A, hipDoubleComplex B, hipDoubleComplex* C, const int Ncn1);
__global__ void multiply_sum_g(hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* C, hipDoubleComplex* D, const int Ncn1);
__global__ void prepare_next_g(hipDoubleComplex* fft_output, hipDoubleComplex* shirin, hipDoubleComplex* V, double* ABL1, hipDoubleComplex epsilon_inv, hipDoubleComplex norm, hipDoubleComplex* shirfn, const int Ncn1);

__global__ void initalize_S_V_gamma_G(hipDoubleComplex* S, hipDoubleComplex* V, hipDoubleComplex* gamma, hipDoubleComplex* G, int Ncn, int Kcn, double dx, double a, hipDoubleComplex S_in, hipDoubleComplex S_out, hipDoubleComplex V_in, hipDoubleComplex V_out, double epsilon, double kf);

int run(int thread_id, int thread_count)
{
  auto begin = chrono::high_resolution_clock::now();

  const int Ncn = 2048;     // 2048;
  const int Kcn = Ncn / 2;  // 1024;
  const int PML = 100;
  const int NFFT = 5;
  const int NDtect = Kcn * Ncn + Kcn + 400;

  double err = 0.0001;

  double dx = 100.0 / 1000000000.0;

  double a = 5.0 / 1000000.0;
  double mu = 1.0;
  double beta = 1.0;
  double Cp = 1.0;
  double I0 = 1.0;

  double f;
  double omega;
  double vf = 1500;
  double vs = 1650;

  double pi = 3.141592653589793;

  double kf;
  double ks;

  double epsilon;

  hipDoubleComplex* shirin;
  hipMalloc((void**)&shirin, sizeof(hipDoubleComplex) * Ncn * Ncn);

  hipDoubleComplex* shirfn;
  hipMalloc((void**)&shirfn, sizeof(hipDoubleComplex) * Ncn * Ncn);

  hipDoubleComplex* V;
  hipMalloc((void**)&V, sizeof(hipDoubleComplex) * Ncn * Ncn);

  hipDoubleComplex* S;
  hipMalloc((void**)&S, sizeof(hipDoubleComplex) * Ncn * Ncn);

  hipDoubleComplex* G;
  hipMalloc((void**)&G, sizeof(hipDoubleComplex) * Ncn * Ncn);

  double* ABL;
  hipMalloc((void**)&ABL, sizeof(double) * Ncn * Ncn);

  hipDoubleComplex* gamma;
  hipMalloc((void**)&gamma, sizeof(hipDoubleComplex) * Ncn * Ncn);

  // hipfftDoubleComplex* fft_input;
  // hipMalloc((void**)&fft_input, sizeof(hipfftDoubleComplex) * Ncn * Ncn);

  // hipfftDoubleComplex* fft_output;
  // hipMalloc((void**)&fft_output, sizeof(hipfftDoubleComplex) * Ncn * Ncn);

  hipfftHandle plan;
  hipfftPlan2d(&plan, Ncn, Ncn, HIPFFT_Z2Z);

  checkCudaErrors(hipGetLastError());

  int BLOCK_SIZE = 16;
  int GRID_SIZE = ceil((float)Ncn / BLOCK_SIZE);

  dim3 grid(GRID_SIZE, GRID_SIZE);
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

  int threads_1d = 256;
  int grid_1d = (Ncn * Ncn + threads_1d - 1) / threads_1d;

  FILE* FID12;
  FID12 = fopen("ShiCBSMultiThreadOMPSingleCell_1650.txt", "w");

  FILE* FID13;
  FID13 = fopen("TimeCBSMultiThreadOMPSingleCell_1650.txt", "w");

  double kkxy;
  int i111;

  time_t t000;
  time(&t000);
  printf("begin : %s", ctime(&t000));

  hipDoubleComplex* temp = (hipDoubleComplex*)malloc(sizeof(hipDoubleComplex) * Ncn * Ncn);
  hipDoubleComplex tempvar;

  hipDoubleComplex* shirfnMid = (hipDoubleComplex*)malloc(sizeof(hipDoubleComplex) * Ncn);
  hipDoubleComplex* shirinMid = (hipDoubleComplex*)malloc(sizeof(hipDoubleComplex) * Ncn);

  for (i111 = thread_id + 1; i111 <= NFFT; i111 += thread_count) {
    auto iter_begin = chrono::high_resolution_clock::now();

    kkxy = 2 * pi * ((double)1 * i111) / (Ncn * dx);
    f = kkxy * vf / (2.0 * pi);
    printf("i111=%i, f=%lf\n", i111, f);

    omega = 2 * pi * f;
    kf = omega / vf;
    ks = omega / vs;
    epsilon = 0.8 * kf * kf;

    // Initialization of S, V, Shi
    hipDoubleComplex S_in = make_hipDoubleComplex(0.0, -(mu * beta * I0 * omega) / Cp);
    hipDoubleComplex S_out = make_hipDoubleComplex(0.0, 0.0);
    hipDoubleComplex V_in = make_hipDoubleComplex(ks * ks - kf * kf, -epsilon);
    hipDoubleComplex V_out = make_hipDoubleComplex(0.0, -epsilon);

    initalize_S_V_gamma_G<<<grid, threads>>>(S, V, gamma, G, Ncn, Kcn, dx, a, S_in, S_out, V_in, V_out, epsilon, kf);

    // hipMemset2D(shirin, sizeof(hipDoubleComplex) * Ncn, 0, Ncn, Ncn);
    // hipMemset2D(shirfn, sizeof(hipDoubleComplex) * Ncn, 0, Ncn, Ncn);

    fftshift_g<<<grid, threads>>>(G, Ncn, Ncn, Kcn);

    hipfftExecZ2Z(plan, S, shirin, HIPFFT_FORWARD);
    multiply_g<<<grid_1d, threads_1d>>>(shirin, G, shirin, Ncn * Ncn);
    hipfftExecZ2Z(plan, shirin, shirin, HIPFFT_BACKWARD);
    multiply_g<<<grid_1d, threads_1d>>>(shirin, gamma, shirin, Ncn * Ncn);
    s_divide_g<<<grid, threads>>>(shirin, make_hipDoubleComplex(Ncn * Ncn, 0), shirin, Ncn);

    absorbinglayer_g<<<grid, threads>>>(ABL, Ncn, Kcn, PML, epsilon, dx);

    checkCudaErrors(hipGetLastError());

    //  ***************************************************
    // Calculation of new field iteratively

    double Error11;
    int i222, ITEmax;

    for (i222 = 0; i222 < 2000; i222++)  //--->// iteration starts
    {
      // multiply_g<<<grid, threads>>>(shirin, V, fft_input, Ncn);
      // sum_g<<<grid, threads>>>(fft_input, S, fft_input, Ncn);
      multiply_sum_g<<<grid_1d, threads_1d>>>(shirin, V, S, shirfn, Ncn * Ncn);

      hipfftExecZ2Z(plan, shirfn, shirfn, HIPFFT_FORWARD);

      multiply_g<<<grid_1d, threads_1d>>>(shirfn, G, shirfn, Ncn * Ncn);

      hipfftExecZ2Z(plan, shirfn, shirfn, HIPFFT_BACKWARD);

      // s_divide_g<<<grid, threads>>>(fft_output, make_hipDoubleComplex(Ncn * Ncn, 0), fft_output, Ncn);
      // sub_g<<<grid, threads>>>(shirin, fft_output, fft_output, Ncn);
      // multiply_g<<<grid, threads>>>(fft_output, V, fft_output, Ncn);
      // s_multiply_g<<<grid, threads>>>(fft_output, make_hipDoubleComplex(0, 1 / epsilon), fft_output, Ncn);
      // sub_g<<<grid, threads>>>(shirin, fft_output, shirfn, Ncn);

      // applyboundarycondition_g<<<grid, threads>>>(shirfn, ABL, Ncn);

      prepare_next_g<<<grid_1d, threads_1d>>>(shirfn, shirin, V, ABL, make_hipDoubleComplex(0, 1 / epsilon), make_hipDoubleComplex(Ncn * Ncn, 0), shirfn, Ncn * Ncn);

      hipMemcpy(shirfnMid, shirfn + Kcn * Ncn, sizeof(hipDoubleComplex) * Ncn, hipMemcpyDeviceToHost);
      hipMemcpy(shirinMid, shirin + Kcn * Ncn, sizeof(hipDoubleComplex) * Ncn, hipMemcpyDeviceToHost);

      ////////
      Error11 = errorcalcualtion(shirfnMid, shirinMid, Ncn, Kcn);  //-->// error calculation

      /////
      if (Error11 <= err) {
        ITEmax = i222;
        break;
      }
      else {
        // hipMemcpy2D(shirin, sizeof(hipDoubleComplex) * Ncn, shirfn, sizeof(hipDoubleComplex) * Ncn, sizeof(hipDoubleComplex) * Ncn, Ncn, hipMemcpyDeviceToDevice);
        swap(shirin, shirfn);
      }

      checkCudaErrors(hipGetLastError());

      // if (i222 % 10 == 0) {
      // printf("Iteration = %d, Error = %lf\n", i222, Error11);
      // }
    }  //*********************----->// iteration stops

    auto iter_end = chrono::high_resolution_clock::now();
    auto iter_time = chrono::duration_cast<chrono::nanoseconds>(iter_end - iter_begin);

    printf("%d: Saturation Iteration: %d, time: %.6f s\n\n", i111, ITEmax, iter_time.count() * 1e-9);

    // For values at NDtect
    // hipMemcpy(&tempvar, shirfn + NDtect, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    // fprintf(FID12, "%lf, %d, %lf, %lf\n", f, ITEmax, tempvar.x, tempvar.y);

    // For all values in the middle row
    // for (int kk = 0; kk < Ncn; kk++) {
    //   fprintf(FID12, "%lf, %d, %lf, %lf\n", f, ITEmax, shirfnMid[kk].x, shirfnMid[kk].y);
    // }
  }

  fclose(FID12);
  fclose(FID13);

  auto end = chrono::high_resolution_clock::now();
  auto total_time = chrono::duration_cast<chrono::nanoseconds>(end - begin);

  printf("Time difference: %.6f s.\n", total_time.count() * 1e-9);

  // Export Shirfn to txt
  // hipMemcpy2D(temp, sizeof(hipDoubleComplex) * Ncn, shirfn, sizeof(hipDoubleComplex) * Ncn, sizeof(hipDoubleComplex) * Ncn, Ncn, hipMemcpyDeviceToHost);

  // FILE *freal, *fimag;
  // char filename[50];
  // snprintf(filename, 50, "../result/Shirfn_%d_real_cuda_7.txt", Ncn);
  // freal = fopen(filename, "w");
  // snprintf(filename, 50, "../result/Shirfn_%d_imag_cuda_7.txt", Ncn);
  // fimag = fopen(filename, "w");

  // for (int i = 0; i < Ncn; i++) {
  //   for (int j = 0; j < Ncn; j++) {
  //     fprintf(freal, "%.15e", temp[i * Ncn + j].x);
  //     fprintf(fimag, "%.15e", temp[i * Ncn + j].y);
  //     if (j < Ncn - 1) {
  //       fprintf(freal, ",");
  //       fprintf(fimag, ",");
  //     }
  //   }
  //   if (i < Ncn - 1) {
  //     fprintf(freal, "\n");
  //     fprintf(fimag, "\n");
  //   }
  // }

  return 0;
}

int main(int argc, char** argv)
{
  int num_threads = 1;
  hipGetDeviceCount(&num_threads);
  printf("Num GPU: %d\n", num_threads);

  for (int i = 0; i < num_threads; i++) {
#pragma omp parallel num_threads(num_threads)
    {
      int thread_id = omp_get_thread_num();
      hipSetDevice(thread_id);
      run(thread_id, num_threads);
    }
  }

  return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void initalize_S_V_gamma_G(hipDoubleComplex* S, hipDoubleComplex* V, hipDoubleComplex* gamma, hipDoubleComplex* G, int Ncn, int Kcn, double dx, double a, hipDoubleComplex S_in, hipDoubleComplex S_out, hipDoubleComplex V_in, hipDoubleComplex V_out, double epsilon, double kf)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int column = blockIdx.x * blockDim.x + threadIdx.x;

  double pi = 3.141592653589793;

  if (row < Ncn && column < Ncn) {
    int index = row * Ncn + column;

    double dist11 = sqrt((float)((row - Kcn) * (row - Kcn) + (column - Kcn) * (column - Kcn)));
    dist11 = dist11 * dx;

    if (dist11 <= a) {
      S[index] = S_in;
      V[index] = V_in;
    }
    else {
      S[index] = S_out;
      V[index] = V_out;
    }

    gamma[index].x = -V[index].y / epsilon;
    gamma[index].y = V[index].x / epsilon;

    double ky = 2 * pi * (row - Kcn) / (Ncn * dx);
    double kx = 2 * pi * (column - Kcn) / (Ncn * dx);

    G[index].x = (kx * kx + ky * ky - kf * kf) / ((kx * kx + ky * ky - kf * kf) * (kx * kx + ky * ky - kf * kf) + epsilon * epsilon);
    G[index].y = epsilon / ((kx * kx + ky * ky - kf * kf) * (kx * kx + ky * ky - kf * kf) + epsilon * epsilon);
  }
}

__global__ void fftshift_g(hipDoubleComplex* M, const int N1, const int N2, const int Kcn)
{
  int i1 = blockIdx.y * blockDim.y + threadIdx.y;
  int j1 = blockIdx.x * blockDim.x + threadIdx.x;

  if (i1 < N1 && j1 < N2) {
    int N11, N22;
    N11 = Kcn;
    N22 = Kcn;

    hipDoubleComplex temp;

    if (i1 < N11 && j1 < N22) {
      long ccn11 = i1 * N2 + j1;
      long ccn22 = (i1 + N11) * N2 + (j1 + N22);
      temp = M[ccn22];
      M[ccn22] = M[ccn11];
      M[ccn11] = temp;
    }
    if (i1 >= N11 && i1 < N1 && j1 < N22) {
      long ccn11 = i1 * N2 + j1;
      long ccn22 = (i1 - N11) * N2 + (j1 + N22);
      temp = M[ccn22];
      M[ccn22] = M[ccn11];
      M[ccn11] = temp;
    }
  }
}

__global__ void absorbinglayer_g(double* mask11, const int Ncn1, const int Kcn1, const int PML1, double epsilon1, double dx1)
{
  int i1 = blockIdx.y * blockDim.y + threadIdx.y;
  int j1 = blockIdx.x * blockDim.x + threadIdx.x;

  int index = i1 * Ncn1 + j1;
  mask11[index] = 1;

  double r = dx1 * (sqrt((float)((i1 - Kcn1) * (i1 - Kcn1) + (j1 - Kcn1) * (j1 - Kcn1))));
  double mask = exp(-r * sqrt(epsilon1));

  if ((i1 < Ncn1 && j1 < Ncn1) && ((i1 < Ncn1 && j1 < PML1)                                               // left
                                   || (i1 < Ncn1 && j1 >= Ncn1 - PML1 && j1 < Ncn1)                       // right
                                   || (i1 < PML1 && j1 >= PML1 && j1 < Ncn1 - PML1)                       // upper
                                   || (i1 >= Ncn1 - PML1 && i1 < Ncn1 && j1 >= PML1 && j1 < Ncn1 - PML1)  // bottom
                                   )) {
    mask11[index] = mask;
  }
}

__global__ void applyboundarycondition_g(hipDoubleComplex* Mfn, double* ABL1, const int Ncn1)
{
  int i1 = blockIdx.y * blockDim.y + threadIdx.y;
  int j1 = blockIdx.x * blockDim.x + threadIdx.x;
  int index = i1 * Ncn1 + j1;

  if (i1 < Ncn1 && j1 < Ncn1) {
    Mfn[index].x = Mfn[index].x * ABL1[index];
    Mfn[index].y = Mfn[index].y * ABL1[index];
  }
}

__global__ void prepare_next_g(hipDoubleComplex* fft_output, hipDoubleComplex* shirin, hipDoubleComplex* V, double* ABL, hipDoubleComplex epsilon_inv, hipDoubleComplex norm, hipDoubleComplex* shirfn, const int N)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < N) {
    shirfn[index] = hipCsub(shirin[index], hipCmul(epsilon_inv, hipCmul(V[index], hipCsub(shirin[index], hipCdiv(fft_output[index], norm)))));
    shirfn[index].x = shirfn[index].x * ABL[index];
    shirfn[index].y = shirfn[index].y * ABL[index];
  }
}

__global__ void multiply_g(hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* C, const int N)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < N) {
    C[index] = hipCmul(A[index], B[index]);
  }
}

__global__ void multiply_sum_g(hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* C, hipDoubleComplex* D, const int N)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < N) {
    D[index] = hipCadd(hipCmul(A[index], B[index]), C[index]);
  }
}

__global__ void s_multiply_g(hipDoubleComplex* A, hipDoubleComplex B, hipDoubleComplex* C, const int Ncn1)
{
  int i1 = blockIdx.y * blockDim.y + threadIdx.y;
  int j1 = blockIdx.x * blockDim.x + threadIdx.x;
  int index = i1 * Ncn1 + j1;

  if (i1 < Ncn1 && j1 < Ncn1) {
    C[index] = hipCmul(A[index], B);
  }
}

__global__ void s_divide_g(hipDoubleComplex* A, hipDoubleComplex B, hipDoubleComplex* C, const int Ncn1)
{
  int i1 = blockIdx.y * blockDim.y + threadIdx.y;
  int j1 = blockIdx.x * blockDim.x + threadIdx.x;
  int index = i1 * Ncn1 + j1;

  if (i1 < Ncn1 && j1 < Ncn1) {
    C[index] = hipCdiv(A[index], B);
  }
}

__global__ void sum_g(hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* C, const int Ncn1)
{
  int i1 = blockIdx.y * blockDim.y + threadIdx.y;
  int j1 = blockIdx.x * blockDim.x + threadIdx.x;
  int index = i1 * Ncn1 + j1;

  if (i1 < Ncn1 && j1 < Ncn1) {
    C[index].x = A[index].x + B[index].x;
    C[index].y = A[index].y + B[index].y;
  }
}

__global__ void sub_g(hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* C, const int Ncn1)
{
  int i1 = blockIdx.y * blockDim.y + threadIdx.y;
  int j1 = blockIdx.x * blockDim.x + threadIdx.x;
  int index = i1 * Ncn1 + j1;

  if (i1 < Ncn1 && j1 < Ncn1) {
    C[index].x = A[index].x - B[index].x;
    C[index].y = A[index].y - B[index].y;
  }
}

double errorcalcualtion(hipDoubleComplex* M, hipDoubleComplex* Q, const int Ncn1, const int Kcn1)
{
  double sum11 = 0.0;
  double sum22 = 0.0;

  for (int i1 = 0; i1 < Ncn1; i1++) {
    sum11 = sum11 + hipCabs(hipCsub(M[i1], Q[i1]));
    sum22 = sum22 + hipCabs(Q[i1]);
  }

  return sum11 / sum22;
}
